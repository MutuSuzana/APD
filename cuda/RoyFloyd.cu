
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <iostream>
#include <fstream>

#define INF 1000000

using namespace std;

__global__ void RoyFloyd(int* matrix, int k, int N)
{
	int i = blockDim.y * blockIdx.y + threadIdx.y;
	int j = blockDim.x * blockIdx.x + threadIdx.x;

	if (matrix[i*N + k] + matrix[k*N + j] < matrix[i*N + j])
		matrix[i*N + j] = matrix[i*N + k] + matrix[k*N + j];
}

int main()
{
	int n;
	int thread_per_block = 1024;

	ifstream input("input.txt");
	ofstream output("output.txt");

	input >> n;

	int num_block = ceil((float)(n*n / (thread_per_block)));

	int *hostMatrix = (int*)malloc(sizeof(int) * n * n);
	int *deviceMatrix;

	for (int i = 0; i < n; i++)
		for (int j = 0; j < n; j++)
		{
			input >> hostMatrix[i + j];
			if (hostMatrix[i + j] == 0)
				hostMatrix[i + j] = INF;
		}

	hipMalloc((void**)&deviceMatrix, n * n * sizeof(int));
	hipMemcpy(deviceMatrix, hostMatrix, n * n * sizeof(int), hipMemcpyHostToDevice);
	
	clock_t t;
	t = clock();

	for (int k = 0; k < n; ++k)
		RoyFloyd <<< num_block, thread_per_block >>>(deviceMatrix, k, n);

	t = clock() - t;
	cout << "Time : " << ((double)t) / CLOCKS_PER_SEC << endl;
	
	hipMemcpy(hostMatrix, deviceMatrix, n * n * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
			if (i == j || hostMatrix[i + j] == INF)
				output << "0 ";
			else
				output << hostMatrix[i + j] << " ";
		output << endl;
	}

	free(hostMatrix);
	hipFree(deviceMatrix);
}